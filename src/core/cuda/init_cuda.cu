/*
 * Copyright (C) 2010-2022 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "init.hpp"
#include "utils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstring>
#include <memory>
#include <string>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

#ifdef CUDA

/** \name minimally required compute capability. */
/**@{*/
static const int computeCapabilityMinMajor = 3;
static const int computeCapabilityMinMinor = 0;
/**@}*/

void cuda_init() { CUDA_CHECK(hipStreamCreate(&stream[0])) }

int cuda_get_n_gpus() {
  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount))
  return deviceCount;
}

bool cuda_check_gpu_compute_capability(int dev) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev))
  return (deviceProp.major < computeCapabilityMinMajor or
          (deviceProp.major == computeCapabilityMinMajor and
           deviceProp.minor < computeCapabilityMinMinor));
}

/**
 * @brief Safely copy the device name and pad the string with null characters.
 */
static void cuda_copy_gpu_name(char *const name, hipDeviceProp_t const &prop) {
  char buffer[256] = {'\0'};
  std::strncpy(buffer, prop.name, 256);
  name[255] = '\0';
  std::strncpy(name, buffer, 256);
}

void cuda_get_gpu_name(int dev, char *const name) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev))
  cuda_copy_gpu_name(name, deviceProp);
}

EspressoGpuDevice cuda_get_device_props(const int dev) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev))
  EspressoGpuDevice device{dev,
                           "",
                           "",
                           -1,
                           deviceProp.major,
                           deviceProp.minor,
                           deviceProp.totalGlobalMem,
                           deviceProp.multiProcessorCount};
  cuda_copy_gpu_name(device.name, deviceProp);
  return device;
}

void cuda_set_device(int dev) {
  CUDA_CHECK(hipSetDevice(dev))
  CUDA_CHECK(hipStreamDestroy(stream[0]))
  CUDA_CHECK(hipStreamCreate(&stream[0]))
}

int cuda_get_device() {
  int dev;
  CUDA_CHECK(hipGetDevice(&dev))
  return dev;
}

bool cuda_test_device_access() {
  auto const deleter = [](int *p) { hipFree(reinterpret_cast<void *>(p)); };
  int *ptr = nullptr;
  int h = 42;
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&ptr), sizeof(int)));
  std::unique_ptr<int, decltype(deleter)> d(ptr, deleter);
  CUDA_CHECK(hipMemcpy(d.get(), &h, sizeof(int), hipMemcpyHostToDevice));
  h = 0;
  CUDA_CHECK(hipMemcpy(&h, d.get(), sizeof(int), hipMemcpyDeviceToHost));
  return h != 42;
}

void cuda_check_device() {
  if (cuda_get_n_gpus() == 0) {
    throw cuda_runtime_error("No GPU was found.");
  }
  auto const devID = cuda_get_device();
  auto const incompatible = cuda_check_gpu_compute_capability(devID);
  auto const communication_failure = cuda_test_device_access();
  if (incompatible or communication_failure) {
    throw cuda_runtime_error("CUDA device " + std::to_string(devID) +
                             " is not capable of running ESPResSo.");
  }
}

#endif /* defined(CUDA) */
